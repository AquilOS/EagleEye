#include "hip/hip_runtime.h"
#include "nvcc_test.cuh"

__global__ void kernel()
{

}
__global__ void kernel(unsigned char* data, int N)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= N)
        return;
    data[tid] *= 2;
}

void run_kernel()
{
    kernel<<<1,1>>>();
}
void run_kernel(unsigned char *data, int pixels, hipStream_t stream)
{
    int threads = 1024;
    int blocks = pixels / 1024;
    kernel<<<blocks, threads, 0, stream>>>(data, pixels);  
}
